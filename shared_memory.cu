//MULTIPLICACIÓN DE MATRICES(APLANADAS)NO CUADRADAS EN C++ y CUDA con tiempo
#include<iostream>
#include<stdio.h>
#include<malloc.h>
#include<hip/hip_runtime.h>
using namespace std; 
#define TILE_WIDTH 32

/*	
* TILE_WIDTH_X debe ser el resultado de columnas de A
* TILE_WIDTH_Y debe ser el resultado de las filas de B
* ya que C[colA][filB] 
* TILE_WIDTH_X es el común entre A y B
*/

__global__ void MultiplicaMatricesCU(int* A,int filA,int colA,int* B,int filB,int colB,int* C){//filC=filA,colC=colB
	
	__shared__ float A_s[TILE_WIDTH][TILE_WIDTH];
	__shared__ float B_s[TILE_WIDTH][TILE_WIDTH];

	// Identifico la fila y la columna de el elemento a trabajar
	int Row = blockIdx.y * TILE_WIDTH + threadIdx.y;//la dimensión del bloque(en shared memory) va a ser el TILE_WIDTH
	int Col = blockIdx.x * TILE_WIDTH + threadIdx.x;

	int suma = 0;
	
	for (int m=0; m < colA/TILE_WIDTH; m++){//¿ hasta el TILE_WIDTH mayor ?

		//sacamos los pedazos con los que vamos a trabajar
		A_s[threadIdx.y][threadIdx.x] = A[Row*colA + ( (m*TILE_WIDTH) + threadIdx.x )];//(Row*colA + k), donde k-> 0..filB (filB = colA)
		// (m*TILE_WIDTH_X) + threadIdx.x aquí nos movemos entre las columnas 
		B_s[threadIdx.y][threadIdx.x] = B[( (m*TILE_WIDTH + threadIdx.y)*colB ) + Col];//(k*colB)+Col, donde k-> 0..filB 
		//(m*TILE_WIDTH_Y + threadIdx.y) se mueve entre las filas 
				
		__syncthreads();//espera a todos los hilos
		
		for (int k=0; k < TILE_WIDTH; ++k) {//es como si se moviera entre 0 y (filB || colA) que son las mismas 
			suma += A_s[threadIdx.y][k] * B_s[k][threadIdx.x];
		}
		__syncthreads();
	}
	C[Row*colB + Col] = suma;//(row*colB)+col
}

__host__ void multiplicaMatrices(int* X,int filX,int colX,int* Y,int filY,int colY,int* Z){
	for(int i=0;i<filX;i++){
		for(int j=0;j<colY;j++){
			int suma=0;
			for(int k=0;k<filY;k++){
				suma=suma+X[(i*colX)+k]*Y[(k*colY)+j];

			}
			Z[(i*colY)+j]=suma;
		}	
	}
}

__host__ void imprime(int* A,int filas, int columnas){//imprime como si fuera una matriz
	for(int i = 0; i < filas; i++){
        	for(int j = 0; j < columnas; j++){
            		cout<<A[(i*columnas)+j]<<" ";
        	}
        cout<<endl;
    }
}	

__host__ void inicializa(int *A,int filas, int columnas){//inicializa arreglos
	for(int i=0;i<filas*columnas;i++){
		A[i]=1;
	}
}

int main(void){

	clock_t startCPU,endCPU,startGPU,endGPU;  
        hipError_t error = hipSuccess;
	int *A,*B,*C; //A[filA][colA],B[filB][colB],C[filA][colB]
	int *d_A,*d_B,*d_C,*h_C;
	int filA=700,colA=1024,filB=1024,colB=1;
	//int filA=5,colA=10,filB=10,colB=1;
	//-------------------------------CPU--------------------------------------------------------------------
	A=(int*)malloc(filA*colA*sizeof(int)); 
	B=(int*)malloc(filB*colB*sizeof(int));
	C=(int*)malloc(filA*colB*sizeof(int));

	inicializa(A,filA,colA);
	inicializa(B,filB,colB);
	
	if(colA==filB){//para que sean multiplicables
		startCPU = clock();		
		multiplicaMatrices(A,filA,colA,B,filB,colB,C);
		endCPU = clock();
		//imprime(C,filA,colB);
	}else{
		cout<<"Error, no se pueden multiplicar"<<endl;
		return 0;
	}

	double time_CPU=((double)(endCPU-startCPU))/CLOCKS_PER_SEC;
	cout<<"El tiempo transcurrido en la CPU fue: "<<time_CPU<<endl;
	//-------------------------------GPU--------------------------------------------------------------------
	h_C=(int*)malloc(filA*colB*sizeof(int));

	startGPU = clock();

	error=hipMalloc((void**)&d_A,filA*colA*sizeof(int));
        if(error != hipSuccess){
            cout<<"Error reservando memoria para d_A"<<endl;
            return -1;
        }
    
	hipMalloc((void**)&d_B,filB*colB*sizeof(int));
        if(error != hipSuccess){
            cout<<"Error reservando memoria para d_B"<<endl;
            return -1;
        }
        
	hipMalloc((void**)&d_C,filA*colB*sizeof(int));	
        if(error != hipSuccess){
            cout<<"Error reservando memoria para d_C"<<endl;
            return -1;
        }
	
	hipMemcpy(d_A,A,filA*colA*sizeof(int),hipMemcpyHostToDevice);//destino d_A y origen A
	hipMemcpy(d_B,B,filB*colB*sizeof(int),hipMemcpyHostToDevice);	

	//Depende directamente de la dimensión de las matrices
	dim3 dimblock(32,32,1);
	dim3 dimGrid(ceil((double)(colA/32)),ceil((double)(filA/32)),1);
	
	MultiplicaMatricesCU<<<dimGrid,dimblock>>>(d_A,filA,colA,d_B,filB,colB,d_C);

	hipDeviceSynchronize();

	hipMemcpy(h_C,d_C,filA*colB*sizeof(int),hipMemcpyDeviceToHost);
	
	endGPU = clock();

	//imprime(h_C,filA,colB);
	double time_GPU=((double)(endGPU-startGPU))/CLOCKS_PER_SEC;
	cout<<"El tiempo transcurrido en la GPU fue: "<<time_GPU<<endl;
	//-----------------------------------------------------------------------------------
	cout<<"El tiempo de aceleramiento fue: "<<time_CPU/time_GPU<<endl;
	free(A);free(B);free(C);free(h_C);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	return 0;
}
