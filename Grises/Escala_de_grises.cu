#include "hip/hip_runtime.h"
//Escala de grises CPU y GPU
#include<iostream>
#include<stdio.h>
#include<malloc.h>
//#include <cv.h>
//#include <highgui.h>
#include<opencv2/opencv.hpp>
using namespace std; 
using namespace cv;

#define RED 2
#define GREEN 1
#define BLUE 0

__host__
void imgTogray(unsigned char *imageInput, int width, int height, unsigned char *imageOutput){
    
    for(int row = 0; row < height; row++){
        for(int col = 0; col < width; col++){
                imageOutput[row*width+col] = imageInput[(row*width+col)*3+RED]*0.299 + imageInput[(row*width+col)*3+GREEN]*0.587 + imageInput[(row*width+col)*3+BLUE]*0.114;
        }
    }
}

__global__ 
void img2gray(unsigned char *imageInput, int width, int height, unsigned char *imageOutput){

    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    if((row < height) && (col < width)){
        
        imageOutput[row*width+col] = imageInput[(row*width+col)*3+RED]*0.299 + imageInput[(row*width+col)*3+GREEN]*0.587 
        + imageInput[(row*width+col)*3+BLUE]*0.114;
    }
}

int main(int argc, char **argv){
    
    hipError_t error = hipSuccess;//Para controlar errores
    unsigned char *h_ImagenInicial, *d_ImagenInicial;
    unsigned char *h_img_gray, *d_img_gray;//Imagen que vamos a pasar a escala de grises
    unsigned char *h_ImagenGrises;
    char* imageName = argv[1];
    Mat image;//Imagen leída
    
    image = imread(imageName, 1);
    
    if(argc !=2 || !image.data){
        printf("No image Data \n");
        return -1;
    }
    
    /*
    //PARA COMPILAR CON EL JUEZ ONLINE
    image = imread("./inputs/img1.jpg", 1);
    if(!image.data){
        printf("No image Data \n");
        return -1;
    }*/
    
    //------------------imágenes--------------------------------
    
    //Sacamos los atributos de la imágen
    Size s = image.size(); 

    int width = s.width;
    int height = s.height;
    int sz = sizeof(unsigned char)*width*height*image.channels();
    int size = sizeof(unsigned char)*width*height;//para la imagen en escala de grises
    
    //Separamos memoria para la imagen inicial en el host y device
    h_ImagenInicial = (unsigned char*)malloc(sz);
    
    error = hipMalloc((void**)&d_ImagenInicial,sz);
    if(error != hipSuccess){
        printf("Error reservando memoria para d_ImagenInicial\n");
        exit(-1);
    }
    
    //Pasamos los datos de la imágen leída 
    h_ImagenInicial = image.data;
    
    //Copiamos los datos al device
    error = hipMemcpy(d_ImagenInicial,h_ImagenInicial,sz, hipMemcpyHostToDevice);
    if(error != hipSuccess){
        printf("Error copiando los datos de h_ImagenInicial a d_ImagenInicial \n");
        exit(-1);
    }
    
    //Separamos memoria para las imágenes a grises en el host y device
    h_img_gray = (unsigned char*)malloc(size);
    
    error = hipMalloc((void**)&d_img_gray,size);
    if(error != hipSuccess){
        printf("Error reservando memoria para d_img_gray\n");
        exit(-1);
    }
    
    dim3 dimBlock(32,32,1);//bloque de 32 x 32 hilos = 1024 hilos
    dim3 dimGrid(ceil(width/float(32)),ceil(height/float(32)),1); 
    img2gray<<<dimGrid,dimBlock>>>(d_ImagenInicial, width, height, d_img_gray);
    hipDeviceSynchronize();
    
    //Copiamos datos de la imágen a escala de grises del device al host
    error = hipMemcpy(h_img_gray,d_img_gray,size, hipMemcpyDeviceToHost);
    if(error != hipSuccess){
        printf("Error copiando los datos de d_img_gray a h_img_gray \n");
        exit(-1);
    }
    
    //Mostramos la imagen en escala de grises de GPU
    Mat resultado_gray_image;
    resultado_gray_image.create(height,width,CV_8UC1);
    resultado_gray_image.data = h_img_gray;
    
    //imshow("Grises",resultado_gray_image);
        
    //imwrite("./outputs/1112786793.png",resultado_gray_image);
    
    
    //Imagen escala de grises CPU
    
    //Separamos memoria para h_ImagenGrises
    
    h_ImagenGrises = (unsigned char*)malloc(size);
    
    imgTogray(h_ImagenInicial, width, height, h_ImagenGrises);
    
    //Mostramos la imagen en escala de grises de CPU
    Mat resultado_gray_imageCPU;
    resultado_gray_imageCPU.create(height,width,CV_8UC1);
    resultado_gray_imageCPU.data = h_ImagenGrises;
    
    //imwrite("./outputs/1112786793.png",resultado_gray_imageCPU);
    imshow("Grises CPU",resultado_gray_imageCPU);
    waitKey(0);
    
    //Liberamos memoria
    free(h_ImagenInicial);free(h_img_gray);
    hipFree(d_ImagenInicial);hipFree(d_img_gray);
    free(h_ImagenGrises);
    
    return 0;
}